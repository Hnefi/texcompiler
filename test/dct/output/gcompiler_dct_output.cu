
#include <hip/hip_runtime.h>
#define COALESCED_NUM 16
#define blockDimX 16
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define globalDimY 1
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_C 2048
#define WIDTH_A 2048
__global__ void dct(float * A, float * B, float * C, int width)
{
	int i;
	float sum;
	__shared__ float shared0[16];
	sum=0;
	{
		shared0[tidx]=A(idy, idx);
	}
	#pragma unroll 
	for (i=0; i<8; i=(i+1))
	{
		float a;
		float b;
		a=shared0[i];
		b=1;
		sum+=(a*b);
	}
	{
		C(idy, idx)=sum;
	}
}
