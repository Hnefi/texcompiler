#include "hip/hip_runtime.h"
#define COALESCED_NUM 16
#define blockDimX 256
#define blockDimY 1
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define idx (blockIdx.x*blockDimX+threadIdx.x)
#define idy (blockIdx.y*blockDimY+threadIdx.y)
#define bidy (blockIdx.y)
#define bidx (blockIdx.x)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define merger_y 8
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define WIDTH_C 2048
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_A (2048+16)
#define A(y,x) A[(y)*WIDTH_A+(x)]
__global__ void demosaic(float * A, float * C, int width)
{
	__shared__ float shared_0[272];
	float temp_0[9];
	float temp_1[9];
	float temp_2[9];
	float temp_3[9];
	float temp_4[9];
	float temp_5[9];
	float temp_6[9];
	float temp_7[9];
	int t_0;
	int t_1;
	int t_2;
	int t_3;
	int t_4;
	int t_5;
	int t_6;
	int t_7;
	int it_1;
	t_0=0;
	t_1=0;
	t_2=0;
	t_3=0;
	t_4=0;
	t_5=0;
	t_6=0;
	t_7=0;
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(3-1)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(3-1)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_0[t_0]=a;
		t_0=(t_0+1);
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(3-2)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(3-2)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_0[t_0]=a;
		temp_1[t_1]=a;
		t_0=(t_0+1);
		t_1=(t_1+1);
	}
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(3-3)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(3-3)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_0[t_0]=a;
		temp_1[t_1]=a;
		temp_2[t_2]=a;
		t_0=(t_0+1);
		t_1=(t_1+1);
		t_2=(t_2+1);
	}
	C(((idy*8)+0), idx)=cal(temp_0);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-1)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-1)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_1[t_1]=a;
		temp_2[t_2]=a;
		temp_3[t_3]=a;
		t_1=(t_1+1);
		t_2=(t_2+1);
		t_3=(t_3+1);
	}
	C(((idy*8)+1), idx)=cal(temp_1);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-2)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-2)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_2[t_2]=a;
		temp_3[t_3]=a;
		temp_4[t_4]=a;
		t_2=(t_2+1);
		t_3=(t_3+1);
		t_4=(t_4+1);
	}
	C(((idy*8)+2), idx)=cal(temp_2);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-3)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-3)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_3[t_3]=a;
		temp_4[t_4]=a;
		temp_5[t_5]=a;
		t_3=(t_3+1);
		t_4=(t_4+1);
		t_5=(t_5+1);
	}
	C(((idy*8)+3), idx)=cal(temp_3);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-4)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-4)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_4[t_4]=a;
		temp_5[t_5]=a;
		temp_6[t_6]=a;
		t_4=(t_4+1);
		t_5=(t_5+1);
		t_6=(t_6+1);
	}
	C(((idy*8)+4), idx)=cal(temp_4);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-5)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-5)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_5[t_5]=a;
		temp_6[t_6]=a;
		temp_7[t_7]=a;
		t_5=(t_5+1);
		t_6=(t_6+1);
		t_7=(t_7+1);
	}
	C(((idy*8)+5), idx)=cal(temp_5);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-6)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-6)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_6[t_6]=a;
		temp_7[t_7]=a;
		t_6=(t_6+1);
		t_7=(t_7+1);
	}
	C(((idy*8)+6), idx)=cal(temp_6);
	__syncthreads();
	if ((tidx<16))
	{
		{
			shared_0[(tidx+0)]=A((((idy*8)+(( - 1)*(0-7)))+16), (idx+(( - 1)*0)));
		}
	}
	{
		shared_0[(tidx+16)]=A((((idy*8)+(( - 1)*(0-7)))+16), ((idx+(( - 1)*0))+16));
	}
	__syncthreads();
	#pragma unroll 
	for (it_1=0; it_1<3; it_1=(it_1+1))
	{
		float a;
		a=shared_0[((tidx+(( - 1)*it_1))+16)];
		temp_7[t_7]=a;
		t_7=(t_7+1);
	}
	C(((idy*8)+7), idx)=cal(temp_7);
	__syncthreads();
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
	{
		
	}
}
