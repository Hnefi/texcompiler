
#include <hip/hip_runtime.h>
#define bidx (blockIdx.x)
#define bidy (blockIdx.y)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define COALESCED_NUM 16
#define blockDimX 16
#define blockDimY 1
#define idx (bidx*blockDimX+tidx)
#define idy (bidy*blockDimY+tidy)
#define merger_y 1
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define WIDTH_C 2048
#define WIDTH_B 2048
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_A 2048
#define A(y,x) A[(y)*WIDTH_A+(x)]
__global__ void matmul(float * A, float * B, float * C, int width, int height)
{
	__shared__ float shared_0[16];
	int i;
	float sum;
	sum=0;
	for (i=0; i<width; i=(i+16))
	{
		int it_1;
		shared_0[(tidx+0)]=A(idy, (i+tidx));
		__syncthreads();
		#pragma unroll 
		for (it_1=0; it_1<16; it_1=(it_1+1))
		{
			float a;
			float b;
			a=shared_0[it_1];
			b=B((it_1+i), idx);
			sum+=(a*b);
		}
		__syncthreads();
	}
	{
		C(idy, idx)=sum;
	}
}
