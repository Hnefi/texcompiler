
#include <hip/hip_runtime.h>
#define bidx (blockIdx.x)
#define bidy (blockIdx.y)
#define tidx (threadIdx.x)
#define tidy (threadIdx.y)
#define gridDimX (gridDim.x)
#define gridDimY (gridDim.y)
#define COALESCED_NUM 16
#define blockDimX 128
#define blockDimY 1
#define idx (bidx*blockDimX+tidx)
#define idy (bidy*blockDimY+tidy)
#define merger_y 32
#define coalesced_idy (bidy/(COALESCED_NUM/(merger_y*blockDimY))*COALESCED_NUM)
#define A(y,x) A[(y)*WIDTH_A+(x)]
#define B(y,x) B[(y)*WIDTH_B+(x)]
#define C(y,x) C[(y)*WIDTH_C+(x)]
#define WIDTH_C 2048
#define WIDTH_B 2048
#define WIDTH_A 2048
__global__ void matmul(float * A, float * B, float * C, int width, int height)
{
	__shared__ float shared_0[16][33];
	int i;
	float sum_0;
	float sum_1;
	float sum_2;
	float sum_3;
	float sum_4;
	float sum_5;
	float sum_6;
	float sum_7;
	float sum_8;
	float sum_9;
	float sum_10;
	float sum_11;
	float sum_12;
	float sum_13;
	float sum_14;
	float sum_15;
	float sum_16;
	float sum_17;
	float sum_18;
	float sum_19;
	float sum_20;
	float sum_21;
	float sum_22;
	float sum_23;
	float sum_24;
	float sum_25;
	float sum_26;
	float sum_27;
	float sum_28;
	float sum_29;
	float sum_30;
	float sum_31;
	sum_0=0;
	sum_1=0;
	sum_2=0;
	sum_3=0;
	sum_4=0;
	sum_5=0;
	sum_6=0;
	sum_7=0;
	sum_8=0;
	sum_9=0;
	sum_10=0;
	sum_11=0;
	sum_12=0;
	sum_13=0;
	sum_14=0;
	sum_15=0;
	sum_16=0;
	sum_17=0;
	sum_18=0;
	sum_19=0;
	sum_20=0;
	sum_21=0;
	sum_22=0;
	sum_23=0;
	sum_24=0;
	sum_25=0;
	sum_26=0;
	sum_27=0;
	sum_28=0;
	sum_29=0;
	sum_30=0;
	sum_31=0;
	for (i=0; i<width; i=(i+16))
	{
		int it_1;
		shared_0[((tidx%16)+0)][(tidx/16)]=A(((((bidy*32)+tidy)+(tidx/16))+0), (i+(tidx%16)));
		shared_0[((tidx%16)+0)][(tidx/16)]=A(((((bidy*32)+tidy)+(tidx/16))+8), (i+(tidx%16)));
		shared_0[((tidx%16)+0)][(tidx/16)]=A(((((bidy*32)+tidy)+(tidx/16))+16), (i+(tidx%16)));
		shared_0[((tidx%16)+0)][(tidx/16)]=A(((((bidy*32)+tidy)+(tidx/16))+24), (i+(tidx%16)));
		__syncthreads();
		#pragma unroll 
		for (it_1=0; it_1<16; it_1=(it_1+1))
		{
			float a_0;
			float a_1;
			float a_2;
			float a_3;
			float a_4;
			float a_5;
			float a_6;
			float a_7;
			float a_8;
			float a_9;
			float a_10;
			float a_11;
			float a_12;
			float a_13;
			float a_14;
			float a_15;
			float a_16;
			float a_17;
			float a_18;
			float a_19;
			float a_20;
			float a_21;
			float a_22;
			float a_23;
			float a_24;
			float a_25;
			float a_26;
			float a_27;
			float a_28;
			float a_29;
			float a_30;
			float a_31;
			float b;
			a_0=shared_0[it_1][0];
			a_1=shared_0[it_1][1];
			a_2=shared_0[it_1][2];
			a_3=shared_0[it_1][3];
			a_4=shared_0[it_1][4];
			a_5=shared_0[it_1][5];
			a_6=shared_0[it_1][6];
			a_7=shared_0[it_1][7];
			a_8=shared_0[it_1][8];
			a_9=shared_0[it_1][9];
			a_10=shared_0[it_1][10];
			a_11=shared_0[it_1][11];
			a_12=shared_0[it_1][12];
			a_13=shared_0[it_1][13];
			a_14=shared_0[it_1][14];
			a_15=shared_0[it_1][15];
			a_16=shared_0[it_1][16];
			a_17=shared_0[it_1][17];
			a_18=shared_0[it_1][18];
			a_19=shared_0[it_1][19];
			a_20=shared_0[it_1][20];
			a_21=shared_0[it_1][21];
			a_22=shared_0[it_1][22];
			a_23=shared_0[it_1][23];
			a_24=shared_0[it_1][24];
			a_25=shared_0[it_1][25];
			a_26=shared_0[it_1][26];
			a_27=shared_0[it_1][27];
			a_28=shared_0[it_1][28];
			a_29=shared_0[it_1][29];
			a_30=shared_0[it_1][30];
			a_31=shared_0[it_1][31];
			b=B((it_1+i), idx);
			sum_0+=(a_0*b);
			sum_1+=(a_1*b);
			sum_2+=(a_2*b);
			sum_3+=(a_3*b);
			sum_4+=(a_4*b);
			sum_5+=(a_5*b);
			sum_6+=(a_6*b);
			sum_7+=(a_7*b);
			sum_8+=(a_8*b);
			sum_9+=(a_9*b);
			sum_10+=(a_10*b);
			sum_11+=(a_11*b);
			sum_12+=(a_12*b);
			sum_13+=(a_13*b);
			sum_14+=(a_14*b);
			sum_15+=(a_15*b);
			sum_16+=(a_16*b);
			sum_17+=(a_17*b);
			sum_18+=(a_18*b);
			sum_19+=(a_19*b);
			sum_20+=(a_20*b);
			sum_21+=(a_21*b);
			sum_22+=(a_22*b);
			sum_23+=(a_23*b);
			sum_24+=(a_24*b);
			sum_25+=(a_25*b);
			sum_26+=(a_26*b);
			sum_27+=(a_27*b);
			sum_28+=(a_28*b);
			sum_29+=(a_29*b);
			sum_30+=(a_30*b);
			sum_31+=(a_31*b);
		}
		__syncthreads();
	}
	{
		C((((bidy*32)+tidy)+0), idx)=sum_0;
	}
	{
		C((((bidy*32)+tidy)+1), idx)=sum_1;
	}
	{
		C((((bidy*32)+tidy)+2), idx)=sum_2;
	}
	{
		C((((bidy*32)+tidy)+3), idx)=sum_3;
	}
	{
		C((((bidy*32)+tidy)+4), idx)=sum_4;
	}
	{
		C((((bidy*32)+tidy)+5), idx)=sum_5;
	}
	{
		C((((bidy*32)+tidy)+6), idx)=sum_6;
	}
	{
		C((((bidy*32)+tidy)+7), idx)=sum_7;
	}
	{
		C((((bidy*32)+tidy)+8), idx)=sum_8;
	}
	{
		C((((bidy*32)+tidy)+9), idx)=sum_9;
	}
	{
		C((((bidy*32)+tidy)+10), idx)=sum_10;
	}
	{
		C((((bidy*32)+tidy)+11), idx)=sum_11;
	}
	{
		C((((bidy*32)+tidy)+12), idx)=sum_12;
	}
	{
		C((((bidy*32)+tidy)+13), idx)=sum_13;
	}
	{
		C((((bidy*32)+tidy)+14), idx)=sum_14;
	}
	{
		C((((bidy*32)+tidy)+15), idx)=sum_15;
	}
	{
		C((((bidy*32)+tidy)+16), idx)=sum_16;
	}
	{
		C((((bidy*32)+tidy)+17), idx)=sum_17;
	}
	{
		C((((bidy*32)+tidy)+18), idx)=sum_18;
	}
	{
		C((((bidy*32)+tidy)+19), idx)=sum_19;
	}
	{
		C((((bidy*32)+tidy)+20), idx)=sum_20;
	}
	{
		C((((bidy*32)+tidy)+21), idx)=sum_21;
	}
	{
		C((((bidy*32)+tidy)+22), idx)=sum_22;
	}
	{
		C((((bidy*32)+tidy)+23), idx)=sum_23;
	}
	{
		C((((bidy*32)+tidy)+24), idx)=sum_24;
	}
	{
		C((((bidy*32)+tidy)+25), idx)=sum_25;
	}
	{
		C((((bidy*32)+tidy)+26), idx)=sum_26;
	}
	{
		C((((bidy*32)+tidy)+27), idx)=sum_27;
	}
	{
		C((((bidy*32)+tidy)+28), idx)=sum_28;
	}
	{
		C((((bidy*32)+tidy)+29), idx)=sum_29;
	}
	{
		C((((bidy*32)+tidy)+30), idx)=sum_30;
	}
	{
		C((((bidy*32)+tidy)+31), idx)=sum_31;
	}
}
