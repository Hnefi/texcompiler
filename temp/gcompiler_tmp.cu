#include "hip/hip_runtime.h"
__global__ void matmul(float * A, float * B, float * C, int width, int height)
{
__shared__ float shared_0[16][17];
int i;
float sum_0;
float sum_1;
float sum_2;
float sum_3;
float sum_4;
float sum_5;
float sum_6;
float sum_7;
float sum_8;
float sum_9;
float sum_10;
float sum_11;
float sum_12;
float sum_13;
float sum_14;
float sum_15;
sum_0=0;
sum_1=0;
sum_2=0;
sum_3=0;
sum_4=0;
sum_5=0;
sum_6=0;
sum_7=0;
sum_8=0;
sum_9=0;
sum_10=0;
sum_11=0;
sum_12=0;
sum_13=0;
sum_14=0;
sum_15=0;
for (i=0; i<width; i=(i+16))
{
int it_1;
shared_0[((tidx%16)+0)][(tidx/16)]=A((((bidy*16)+tidy)+(tidx/16)), (i+(tidx%16)));
__syncthreads();
#pragma unroll

for (it_1=0; it_1<16; it_1=(it_1+1))
{
float a_0;
float a_1;
float a_2;
float a_3;
float a_4;
float a_5;
float a_6;
float a_7;
float a_8;
float a_9;
float a_10;
float a_11;
float a_12;
float a_13;
float a_14;
float a_15;
float b;
a_0=shared_0[it_1][0];
a_1=shared_0[it_1][1];
a_2=shared_0[it_1][2];
a_3=shared_0[it_1][3];
a_4=shared_0[it_1][4];
a_5=shared_0[it_1][5];
a_6=shared_0[it_1][6];
a_7=shared_0[it_1][7];
a_8=shared_0[it_1][8];
a_9=shared_0[it_1][9];
a_10=shared_0[it_1][10];
a_11=shared_0[it_1][11];
a_12=shared_0[it_1][12];
a_13=shared_0[it_1][13];
a_14=shared_0[it_1][14];
a_15=shared_0[it_1][15];
b=B((it_1+i), idx);
sum_0+=(a_0*b);
sum_1+=(a_1*b);
sum_2+=(a_2*b);
sum_3+=(a_3*b);
sum_4+=(a_4*b);
sum_5+=(a_5*b);
sum_6+=(a_6*b);
sum_7+=(a_7*b);
sum_8+=(a_8*b);
sum_9+=(a_9*b);
sum_10+=(a_10*b);
sum_11+=(a_11*b);
sum_12+=(a_12*b);
sum_13+=(a_13*b);
sum_14+=(a_14*b);
sum_15+=(a_15*b);
}
__syncthreads();
}
int it_0;
for (it_0=0; it_0<1; it_0=(it_0+1))
{
C((((bidy*16)+tidy)+0), idx)=sum_0;
}
int it_2;
for (it_2=0; it_2<1; it_2=(it_2+1))
{
C((((bidy*16)+tidy)+1), idx)=sum_1;
}
int it_3;
for (it_3=0; it_3<1; it_3=(it_3+1))
{
C((((bidy*16)+tidy)+2), idx)=sum_2;
}
int it_4;
for (it_4=0; it_4<1; it_4=(it_4+1))
{
C((((bidy*16)+tidy)+3), idx)=sum_3;
}
int it_5;
for (it_5=0; it_5<1; it_5=(it_5+1))
{
C((((bidy*16)+tidy)+4), idx)=sum_4;
}
int it_6;
for (it_6=0; it_6<1; it_6=(it_6+1))
{
C((((bidy*16)+tidy)+5), idx)=sum_5;
}
int it_7;
for (it_7=0; it_7<1; it_7=(it_7+1))
{
C((((bidy*16)+tidy)+6), idx)=sum_6;
}
int it_8;
for (it_8=0; it_8<1; it_8=(it_8+1))
{
C((((bidy*16)+tidy)+7), idx)=sum_7;
}
int it_9;
for (it_9=0; it_9<1; it_9=(it_9+1))
{
C((((bidy*16)+tidy)+8), idx)=sum_8;
}
int it_10;
for (it_10=0; it_10<1; it_10=(it_10+1))
{
C((((bidy*16)+tidy)+9), idx)=sum_9;
}
int it_11;
for (it_11=0; it_11<1; it_11=(it_11+1))
{
C((((bidy*16)+tidy)+10), idx)=sum_10;
}
int it_12;
for (it_12=0; it_12<1; it_12=(it_12+1))
{
C((((bidy*16)+tidy)+11), idx)=sum_11;
}
int it_13;
for (it_13=0; it_13<1; it_13=(it_13+1))
{
C((((bidy*16)+tidy)+12), idx)=sum_12;
}
int it_14;
for (it_14=0; it_14<1; it_14=(it_14+1))
{
C((((bidy*16)+tidy)+13), idx)=sum_13;
}
int it_15;
for (it_15=0; it_15<1; it_15=(it_15+1))
{
C((((bidy*16)+tidy)+14), idx)=sum_14;
}
int it_16;
for (it_16=0; it_16<1; it_16=(it_16+1))
{
C((((bidy*16)+tidy)+15), idx)=sum_15;
}
}

